#include "hip/hip_runtime.h"

/* dof_gpu.cu.
 * 
 * This file contains the definition of the CUDA functions ,
 * for rendering depth of field, based on Gaussian blurring
 * using separable convolution, with depth-dependent kernel size.
 * Separable convolution is based on convolution CUDA Sample with kernel-size adaptation
*/


#include "dof_gpu.h"
#include <iostream>
#include <assert.h>

 
__constant__ float c_kernel[KERNEL_RADIUS * (KERNEL_RADIUS + 2)];

extern "C" void copyKernel(float *kernel_coefficients, int kernel_index){
	int kernel_radius = kernel_index + 1;
	hipMemcpyToSymbol(HIP_SYMBOL(
        c_kernel), 
        kernel_coefficients, 
        KERNEL_LENGTH_X(kernel_radius) * sizeof(float),
        kernel_index * (kernel_index + 2) * sizeof(float));
}

 
 

__global__ void _k_normalizeDepth(float* depth,float* depth_norm, unsigned int step,float min_distance, float max_distance,unsigned int width, unsigned height)
{
	uint32_t x_local = blockIdx.x*blockDim.x + threadIdx.x;
	uint32_t y_local = blockIdx.y*blockDim.y + threadIdx.y;

	if (x_local >= width || y_local >= height) return;

	float depth_world = depth[x_local + y_local *step];
	float depth_normalized = (max_distance - depth_world) / (max_distance - min_distance);



	if (depth_normalized < 0.f) depth_normalized = 0.f;
	if (depth_normalized > 1.f) depth_normalized = 1.f;

	if(isfinite(depth_normalized))
		depth_norm[x_local + y_local *step] = depth_normalized;



}


extern "C" void normalizeDepth(float* depth, float* depth_out, unsigned int step, float min_distance, float max_distance, unsigned int width, unsigned height)
{
	dim3 dimGrid, dimBlock;

	dimBlock.x = 32;
	dimBlock.y = 8;

	dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;


	_k_normalizeDepth << <dimGrid, dimBlock, 0 >> > (depth, depth_out, step, min_distance, max_distance, width,height);


}







////////////////////////////////////////////////////////////////////////////////
// Convolution kernel storage
////////////////////////////////////////////////////////////////////////////////
//__constant__ float c_Kernel[KERNEL_LENGTH];

//__constant__ float c_kernel[NUM_KERNELS * (NUM_KERNELS + 2)];
extern "C" void setConvolutionKernel(float *h_Kernel)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_kernel), h_Kernel, KERNEL_LENGTH * sizeof(float));
}


////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
#define   ROWS_BLOCKDIM_X 32
#define   ROWS_BLOCKDIM_Y 4
#define ROWS_RESULT_STEPS 8
#define   ROWS_HALO_STEPS 1

__global__ void convolutionRowsKernel(
	uchar4 *d_Dst,
	uchar4 *d_Src,
	float* depth,
	int imageW,
	int imageH,
	int pitch,
	int pitch_depth,
	float focus_depth
)
{
	__shared__ uchar4 s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

	//Offset to the left halo edge
	const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
	const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

	d_Src += baseY * pitch + baseX;
	d_Dst += baseY * pitch + baseX;
	depth += baseY * pitch_depth + baseX;



	uchar4 reset = make_uchar4(0, 0, 0, 0);
	//Load main data
#pragma unroll

	for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
	{
		s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
	}

	//Load left halo
#pragma unroll

	for (int i = 0; i < ROWS_HALO_STEPS; i++)
	{
		s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : reset;
	}

	//Load right halo
#pragma unroll

	for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
	{
		s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : reset;
	}

	//Compute and store results
	__syncthreads();
#pragma unroll

	for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
	{
		float4 sum = make_float4(0,0,0,0);

#pragma unroll
		int kernel_radius = (int)floor((KERNEL_RADIUS)*fabs(depth[i * ROWS_BLOCKDIM_X] - focus_depth));
		int kernel_start = kernel_radius * kernel_radius - 1;
		int kernel_mid = kernel_start + kernel_radius;

		if (kernel_radius > 0)
		{
			for (int j = -kernel_radius; j <= kernel_radius; ++j)
			{
				sum.x += c_kernel[kernel_mid + j] * (float)s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j].x;
				sum.y += c_kernel[kernel_mid + j] * (float)s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j].y;
				sum.z += c_kernel[kernel_mid + j] * (float)s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j].z;
			}
		}
		else
		{
			sum.x = (float)s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X ].x;
			sum.y =(float)s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X ].y;
			sum.z =(float)s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X ].z;
		}


		//float depth_8U = depth[i * ROWS_BLOCKDIM_X] * 255.f;
		d_Dst[i * ROWS_BLOCKDIM_X] =  make_uchar4(sum.x,sum.y,sum.z, 255);
 
	}
}

extern "C" void convolutionRowsGPU(
	uchar4 *d_Dst,
	uchar4 *d_Src,
	float* i_depth,
	int imageW,
	int imageH,
	int depth_pitch,
	float focus_point
)
{
	assert(ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS);
	assert(imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0);
	assert(imageH % ROWS_BLOCKDIM_Y == 0);

	dim3 blocks(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), imageH / ROWS_BLOCKDIM_Y);
	dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);

	convolutionRowsKernel << <blocks, threads >> >(
		d_Dst,
		d_Src,
		i_depth,
		imageW,
		imageH,
		imageW,
		depth_pitch,
		focus_point
		);

}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
#define   COLUMNS_BLOCKDIM_X 16
#define   COLUMNS_BLOCKDIM_Y 8
#define COLUMNS_RESULT_STEPS 2
#define   COLUMNS_HALO_STEPS 4

__global__ void convolutionColumnsKernel(
	uchar4 *d_Dst,
	uchar4 *d_Src,
	float* depth,
	int imageW,
	int imageH,
	int pitch,
	int pitch_depth,
	float focus_depth
)
{
	__shared__ uchar4 s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];
	uchar4 reset = make_uchar4(0, 0, 0, 0);
	//Offset to the upper halo edge
	const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
	const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
	d_Src += baseY * pitch + baseX;
	d_Dst += baseY * pitch + baseX;
	depth += baseY * pitch_depth + baseX;


	//Main data
#pragma unroll

	for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
	{
		s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_Src[i * COLUMNS_BLOCKDIM_Y * pitch];
	}

	//Upper halo
#pragma unroll

	for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
	{
		s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : reset;
	}

	//Lower halo
#pragma unroll

	for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
	{
		s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : reset;
	}

	//Compute and store results
	__syncthreads();
#pragma unroll

	for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
	{
		float4 sum = make_float4(0,0,0,0);
		int kernel_radius =  (int)floor((KERNEL_RADIUS)*fabs(depth[i * COLUMNS_BLOCKDIM_Y * pitch] - focus_depth));
		int kernel_start = kernel_radius * kernel_radius - 1;
		int kernel_mid = kernel_start + kernel_radius;
		
		if (kernel_radius > 0)
		{
			for (int j = -kernel_radius; j <= kernel_radius; ++j)
			{
					sum.x += c_kernel[kernel_mid + j] * (float)s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j].z;
					sum.y += c_kernel[kernel_mid + j] * (float)s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j].y;
					sum.z += c_kernel[kernel_mid + j] * (float)s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j].x;
			}
		}
		else
		{
			sum.x = (float)s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y].z;
			sum.y = (float)s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y ].y;
			sum.z = (float)s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y ].x;
		}
	 
		//float depth_8U = depth[i * COLUMNS_BLOCKDIM_Y * pitch] * 255.f;
		d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = make_uchar4(sum.x, sum.y, sum.z, 255);
	}
}

extern "C" void convolutionColumnsGPU(
	uchar4 *d_Dst,
	uchar4 *d_Src,
	float* i_depth,
	int imageW,
	int imageH,
	int depth_pitch, 
	float focus_point
)
{
	assert(COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= KERNEL_RADIUS);
	assert(imageW % COLUMNS_BLOCKDIM_X == 0);
	assert(imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0);

	dim3 blocks(imageW / COLUMNS_BLOCKDIM_X, imageH / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
	dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);


	convolutionColumnsKernel << <blocks, threads >> >(
		d_Dst,
		d_Src,
		i_depth,
		imageW,
		imageH,
		imageW,
		depth_pitch,
		focus_point
		);

}


